/* ###########################################################################################################################
## Organization         : The University of Arizona
##                      :
## File name            : GaB.c
## Language             : C (ANSI)
## Short description    : Gallager-B Hard decision Bit-Flipping algorithm
##                      :
##                      :
##                      :
## History              : Modified 19/01/2016, Created by Burak UNAL
##                      :
## COPYRIGHT            : burak@email.arizona.edu
## ######################################################################################################################## */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdio.h>
#include <unistd.h>

//#####################################################################################################
__global__ void DataPassGB(int *VtoC, int *CtoV, int *Receivedword, int *Interleaver,int ColumnDegree,int N,int NbBranch, int iter)
{
	int t,numB,n,buf;
	int Global;
	numB=0;
	
        n = threadIdx.x + blockIdx.x*blockDim.x;
        numB = ColumnDegree * n;
    if (n < N) {
        if (iter == 0) {
              for (t=0;t<ColumnDegree;t++)     
               VtoC[Interleaver[numB+t]]=Receivedword[n];
        } else {
        
		       //Global=(Amplitude)*(1-2*ReceivedSymbol[n]);
		       Global=(1-2*Receivedword[n]); 
		       //Global=(1-2*(Decide[n] + Receivedword[n])); //Decide[n]^Receivedword[n];
		       for (t=0;t<ColumnDegree;t++) Global+=(-2)*CtoV[Interleaver[numB+t]]+1;

		       for (t=0;t<ColumnDegree;t++)
		       {
		            buf=Global-((-2)*CtoV[Interleaver[numB+t]]+1);
		            if (buf<0)  VtoC[Interleaver[numB+t]]= 1; //else VtoC[Interleaver[numB+t]]= 1;
		            else if (buf>0) VtoC[Interleaver[numB+t]]= 0; //else VtoC[Interleaver[numB+t]]= 1;
		            else  VtoC[Interleaver[numB+t]]=Receivedword[n];
		        }
           }
     }
	
}
//#####################################################################################################


//##################################################################################################
__global__ void CheckPassGB(int *CtoV,int *VtoC,int M,int NbBranch,int RowDegree)
{
   int t,numB=0,m,signe;
   m = threadIdx.x + blockIdx.x*blockDim.x;
   numB= RowDegree * m;
     if (m < M) {
		signe=0;for (t=0;t<RowDegree;t++) signe^=VtoC[numB+t];
	    for (t=0;t<RowDegree;t++) 	CtoV[numB+t]=signe^VtoC[numB+t];
    }

}
//#####################################################################################################
__global__ void APP_GB(int *Decide,int *CtoV,int *Receivedword,int *Interleaver,int ColumnDegree,int N,int M,int NbBranch)
{
   	int t,numB,n;
	int Global;
    n = threadIdx.x + blockIdx.x*blockDim.x;
	numB=ColumnDegree * n;
    

    if (n < N) {
		Global=(1-2*Receivedword[n]);
		for (t=0;t<ColumnDegree;t++) Global+=(-2)*CtoV[Interleaver[numB+t]]+1;
        if(Global>0) Decide[n]= 0;
        else if (Global<0) Decide[n]= 1;
        else  Decide[n]=Receivedword[n];
    }

}
//#####################################################################################################
__global__ void ComputeSyndrome(int *Decide,int *Mat,int RowDegree,int M, int *Dev_Syndrome)
{
	int Synd,l;
    //This needs reduction function 
    __shared__ int sh_Synd[648];
    
     int n = threadIdx.x + blockIdx.x*blockDim.x;
     int thd_id = threadIdx.x;

     if(n ==0 ) *Dev_Syndrome = 1;
     
     for (l=0;l<RowDegree;l++)Synd=Synd^Decide[Mat[n*8 + l]];    

     if (n < M) sh_Synd[thd_id] = Synd; 
     __syncthreads();
     
    //Reduce to a single value 
    for(int stride = blockDim.x/2 ; stride > 0; stride = stride/2) {
     sh_Synd[thd_id] = sh_Synd[thd_id] | sh_Synd[thd_id + stride];
     __syncthreads();
     }
    
     if (thd_id == 0 ) atomicMin(Dev_Syndrome, (1 - sh_Synd[0])); 

}

